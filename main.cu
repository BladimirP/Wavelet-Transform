#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

/*
    Args:
        lectura :
*/
__global__ void wavelet_transform(int *lectura)
{
    
}


int main(int argc, char *argv[])
{
    // ########## VALIDACION ##########
    if (argc < 2)
    {
        printf("Usage: ./prog <rt> <bl>\n"
            "rt (route)     <STR>\n"
            "bl (blockSize) <INT>");
        return 1;
    }

    // ########## ARGUMENTOS ##########
    String routeMP3 = argv[1];
    int bl = atoi(argv[2]);

    // ########## MEMORIA CPU ##########
    int *outputCPU = (int *)malloc(sizeof(int) * width * height);
    initArray(outputCPU, width, height);
    
    // ########## MEMORIA GPU ##########
    int *outputGPU;
    hipMalloc((void **)&outputGPU, sizeof(int) * width * height);

    // ########## CPY CPU~GPU ##########
    hipMemcpy(outputGPU, outputCPU, sizeof(int) * width * height, hipMemcpyHostToDevice);

    // ########## GRID & BLOCKS ##########
    dim3 gridSize(4, 1);    // Grid con 4 bloques
    dim3 blockSize(1, 1);   // Bloque con 1 hilo

    // ########## KERNEL ##########
    wavelet_transform<<<gridSize, blockSize>>>(outputGPU);
    
    // ########## CPY GPU~CPU ##########
    hipMemcpy(outputCPU, outputGPU, sizeof(int) * n * n, hipMemcpyDeviceToHost);

    // Display Informacion


    // Liberando memoria
    free(outputCPU);
    hipFree(outputGPU);

    return 0;
}
