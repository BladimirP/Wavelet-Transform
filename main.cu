#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

/*
    Args:
        lectura :
*/
__global__ void wavelet_transform(int *lectura)
{
    
}


int main(int argc, char *argv[])
{
    if (argc < 2)
    {
        printf("Usage: ./marianisilver <./.../archivo.mp3>\n");
        return 1;
    }

    // Obteniendo argumentos de ejecucion
    String routeMP3 = argv[1];


    printf("Parameters ~ routeMP3 = %s\n", routeMP3);

    // Reservando Memoria CPU
    int *outputCPU = (int *)malloc(sizeof(int) * width * height);
    initArray(outputCPU, width, height);
    
    // Reservando memoria GPU
    int *outputGPU;
    hipMalloc((void **)&outputGPU, sizeof(int) * width * height);

    // Copiar el array desde la CPU a CPU
    hipMemcpy(outputGPU, outputCPU, sizeof(int) * width * height, hipMemcpyHostToDevice);

    // Configuracion del Kernel
    dim3 gridSize(4, 1);    // Grid con 4 bloques
    dim3 blockSize(1, 1);   // Bloque con 1 hilo

    // LLamada al kernel wavelet_transform
    wavelet_transform<<<gridSize, blockSize>>>(outputGPU);
    
    // Copiando resultados de GPU a CPU
    hipMemcpy(outputCPU, outputGPU, sizeof(int) * n * n, hipMemcpyDeviceToHost);

    // Display Informacion


    // Liberando memoria
    free(outputCPU);
    hipFree(outputGPU);

    return 0;
}
